#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>
#include "hip/hip_runtime.h"


const int N = 1024 * 1024 * 64 * 16;

const int grid_size = 112;
const int block_size = 1024;

static long long ustime(void) {
	struct timeval tv;
	long long ust;
	gettimeofday(&tv, NULL);
	ust = ((long)tv.tv_sec)*1000000;
	ust += tv.tv_usec;
	return ust;
}

double getPi(int n) {
	double s = 0;
		for (int i = 0; i < n; i++) {
		double t = (2 * i + 1) / (n * 2.0);
		s += 4 / (1 + t * t);
	}
	return s / n;
}

__global__ void reducePi(double *sum) {
	__shared__ double cache[block_size];
	int tid = blockDim.x * blockIdx.x + threadIdx.x;
	int cacheIdx = threadIdx.x;

	double x, t = 0;
	while (tid < N) {
		x = (2 * tid + 1) / (N * 2.0);
		t += 4 / (1 + x * x);
		tid += blockDim.x * gridDim.x;
	}
	cache[cacheIdx] = t;
	__syncthreads();

	for (int i = (blockDim.x >> 1); i; i >>= 1) {
		if (cacheIdx < i) {
			cache[cacheIdx] += cache[cacheIdx + i];
		}
		__syncthreads();
	}
	if (!cacheIdx) {
		sum[blockIdx.x] = cache[0];
	}
}

__global__ void sumUp(double *sum, double *s) {
	__shared__ double cache[grid_size];
	int tid = threadIdx.x;
	cache[tid] = sum[tid];
	__syncthreads();
	for (int i = (blockDim.x >> 1); i; i >>= 1) {
		if (tid < i) {
			cache[tid] += cache[tid + i];
		}
		__syncthreads();
	}
	if (!tid) {
		*s = cache[0];
	}
}

int main() {
	long long op, ed, tcpu, tgpu;
	double sum, ans;

	double *dev_s_sum;
	double *dev_sum;

	//int grid_size, grid_min_size, block_size;

	//int gs, bs;
	//cudaOccupancyMaxPotentialBlockSize(&gs, &bs, reducePi, 0, N);
	//printf("%d %d\n", gs, bs);
	//grid_size = (N - 1) / block_size + 1;

	hipMalloc((void **) &dev_s_sum, grid_size * sizeof(double));
	hipMalloc((void **) &dev_sum, sizeof(double));

	op = ustime();
	reducePi <<<grid_size, block_size>>>(dev_s_sum);
	sumUp <<<1, 64>>>(dev_s_sum, dev_sum);
	ed = ustime();
	tgpu = ed - op;

	hipMemcpy(&sum, dev_sum, sizeof(double), hipMemcpyDeviceToHost);
	ans = sum / double(N);
	printf("GPU: Pi: %.12lf Time(ms): %f\n", ans, tgpu / 1000.0);

	hipFree(dev_s_sum);
	hipFree(dev_sum);

	op = ustime();
	ans = getPi(N);
	ed = ustime();
	tcpu = ed - op;
	printf("CPU: Pi: %.12f Time(ms): %f\n", ans, tcpu / 1000.0);
	printf("Speed up: %.2f\n", tcpu / (double) tgpu);
}
